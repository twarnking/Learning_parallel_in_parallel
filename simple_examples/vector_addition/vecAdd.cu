#include <iostream>
#include <hip/hip_runtime.h>


int main(){
  int n = 1024;
  float* h_a, * h_b, * h_c;
  float* d_a, * d_b, *d_c;

  // create the host memory
  h_a = new float[n];
  h_b = new float[n];
  h_c = new float[n];


  // create the device memory
  hipMalloc(&d_a, n * sizeof(float));
  hipMalloc(&d_b, n * sizeof(float));
  

}
